#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#define cudaCheckErrors(msg) \
do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg, hipGetErrorString(__err), __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
} while (0)

__global__ void kernel(int *output, const int *input) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    output[id] = input[id];
}

void launch(int gridSizeX, int blockSizeX, int gridSizeY = 1, int blockSizeY = 1, int gridSizeZ = 1, int blockSizeZ = 1) {
    dim3 blockSize(blockSizeX, blockSizeY, blockSizeZ);
    dim3 gridSize(gridSizeX, gridSizeY, gridSizeZ);
    kernel<<<gridSize, blockSize>>>(outputPtr, inputPtr);
}

int main() {
    int N = 1024;
    int *outputPtr, *inputPtr;
    int hostInput[N]; // Assuming hostInput is defined and initialized

    hipMalloc(&outputPtr, sizeof(int) * N);
    hipMalloc(&inputPtr, sizeof(int) * N);
    hipMemcpy(inputPtr, hostInput, sizeof(int) * N, hipMemcpyHostToDevice);

    launch(4, 1024);
    cudaCheckErrors("kernel launch failed");

    launch(4, 32, 4, 32);
    cudaCheckErrors("kernel launch failed");

    launch(4, 16, 4, 16, 4, 4);
    cudaCheckErrors("kernel launch failed");

    hipFree(outputPtr);
    hipFree(inputPtr);
}
